#include "hip/hip_runtime.h"
//===========================================================================//
// File Name:   simplex_gpu_kernels.cpp
// Authors:     Cody Cziesler, Praneeth Pulusani
//
// Description: The simplex algorithm implemented on a GPU
//
//===========================================================================//

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <iostream>
#include "stdio.h" 

#include "common_gpu.h"
#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ < 200)
   #define printf(f, ...) ((void)(f, __VA_ARGS__),0)
#endif
//===========================================================================//
// normalize_kernel
//
// Scale each column of the pivot row by input "scale"
//
// @param  arr        - the 1D array representation of the matrix
// @param  scale      - arr[pivotRow * width + pivotColumn]
// @param  height     - the height of arr
//===========================================================================//
__global__ void normalize_kernel (float* arr, float scale, int height, int width, int pivotRow) {
  //printf("kernel\n");
	
  // Indexing
  int yId = threadIdx.y + (blockIdx.y * blockDim.y); // Row

  // Skip if not within size
  if (yId >= width) {
    return;
  }

  // Scale the row by "scale"
  arr[INDEX(yId, pivotRow)] /= scale;

}

//===========================================================================//
// row_reduce_kernel
//
// Perform row reduction on each row
//
// @param  arr        - the 1D array representation of the matrix
// @param  divider    - arr[INDEX(pivotColumn, PivotRow)]
// @param  height     - the height of arr
//===========================================================================//
__global__ void row_reduce_kernel (float* arr, int height, int width, int pivotRow, int pivotColumn) {

  // Indexing
  int xId = threadIdx.x + (blockIdx.x * blockDim.x); // Column
  int yId = threadIdx.y + (blockIdx.y * blockDim.y); // Row
  float scale;

  // Skip if not within size, or on pivotRow
  if ( (yId >= height) || (xId >= width) || (yId == pivotRow) ) {
    return;
  }
  
  // Find the scale (will be different for each row
  scale = arr[INDEX(pivotColumn, yId)];

  float divider = arr[INDEX(pivotColumn, pivotRow)];
  // Row Reduction for each pixel
  //printf("x: %d, y: %d, value: %f = %f- %f* (%f/%f)\n",xId,yId,arr[INDEX(xId, yId)],arr[INDEX(xId, yId)],arr[INDEX(xId, pivotRow)],scale,divider);
  arr[INDEX(xId, yId)] -= (arr[INDEX(xId, pivotRow)] * (scale / divider));

  //printf("pivotColumn: %d, width: %d, height: %d, gausselim col:%d,row:%d,pivot row:%d,scale/div:%f\n",pivotColumn, width,height,xId,yId,pivotRow,scale/divider);


}

//===========================================================================//
// simplex_gpu
//
// Puts the whole Simplex Algorithm together
//
// @param  arr        - the 1D array representation of the matrix
// @param  width      - the width of arr
// @param  height     - the height of arr
//
// @return num_iterations - the number of iterations it took to complete
//===========================================================================//
int simplex_gpu (float *arr, int width, int height) {
	cout<<"hello there"<<endl;
  DBGPRINT("Simplex GPU function entered ");
  // A status flag
  hipError_t status;

  int num_iterations = 0;
  float scale;
  int pivotRow, pivotColumn;
  int tile_size = 512;
  int size = width * height;


  // Number of bytes in the matrix. 
  int bytes = size * sizeof(float); 
  // Pointers to the device arrays 
  float *arr_d; 
  // Allocate memory on the device to store each matrix 
  hipMalloc((void**) &arr_d, bytes); 
  // Copy the host input data to the device 
  hipMemcpy(arr_d, arr, bytes, hipMemcpyHostToDevice); 

  int dim_size = (int)ceil((float)size / (float)tile_size);

  // Row Reduction Dimensions (2D)
  dim3 dimGridRR(tile_size, tile_size);
  dim3 dimBlockRR(dim_size, dim_size);

  // Normalize Dimensions (1D)
  dim3 dimGridN(1, tile_size);
  dim3 dimBlockN(1, dim_size);

  /////////////////////////////////////////////////
  // Repeat until the bottom row is all positive //
  /////////////////////////////////////////////////
  while (!is_indicator_positive_gpu (arr, width, height)) {
    cout<<"hello there1"<<endl;
    // If number of iterations exceed the threshold, no solutions were found
    if (num_iterations > MAX_ITER) {
      return num_iterations;
    }

    DBGPRINT("Iteration " << num_iterations);

    // Do the gaussian elimination part
    pivotColumn = get_pivot_column_index_gpu (arr,width,height);
    pivotRow    = get_pivot_row_index_gpu (arr,width,height,pivotColumn);

	DBGPRINT("Iteration " );
	cout<<"hello there normalization"<<endl;
    // Normalization
    scale = arr[INDEX(pivotColumn, pivotRow)];
    normalize_kernel<<<dimGridN, dimBlockN>>>(arr_d, scale, height, width, pivotRow);
	hipDeviceSynchronize();

    // Check for CUDA errors
    status = hipGetLastError();
    if (status != hipSuccess) {
      cout << "Kernel failed: " << hipGetErrorString(status) << endl;
      return -1;
    }
	cout<<"hello there row reduction"<<endl;
	DBGPRINT("Before row reduction ");
    // Row reduction
    row_reduce_kernel<<<dimGridRR, dimBlockRR>>>(arr_d, height, width, pivotRow, pivotColumn);
	hipDeviceSynchronize();

	//return 0;
    // Check for CUDA errors
    status = hipGetLastError();
    if (status != hipSuccess) {
      cout << "Kernel failed: " << hipGetErrorString(status) << endl;
      return -1;
    }

    // Increment the number of iterations
    num_iterations++;
    #ifdef DEBUG
     // print_matrix_gpu (arr, width, height);
    #endif
	 // Copy the host input data to the device 
	 hipMemcpy(arr, arr_d, bytes, hipMemcpyDeviceToHost); 
	  // print_matrix_gpu(arr,width,height);
  }
print_matrix_gpu(arr,width,height);

  cout << "Printing GPU solution\n";

  return num_iterations;

}
